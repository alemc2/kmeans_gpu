#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <float.h>

#include "kmeans.h"

#define BLOCK_SZ_CNT_ASS 256
#define BLOCK_SZ_CNT_ADJ 256

//Define some counters on device for access to changes
__device__ uint32_t mem_change_ctr;
//Define these constant variables that are going to be that way for the entire
//experiment
__constant__ uint32_t num_features;
__constant__ uint32_t num_samples;
__constant__ uint32_t num_clusters;
__constant__ uint32_t shmem_size;
__constant__ int d_debug;

// https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
__device__ __forceinline__ uint32_t atomicAggInc(uint32_t *ctr) {
  int mask = __ballot(1);
  int leader = __ffs(mask) - 1;
  uint32_t res;
  if ((threadIdx.x % 32) == leader) {
    res = atomicAdd(ctr, __popc(mask));
  }
  res = __shfl(res, leader);
  return res + __popc(mask & ((1 << (threadIdx.x % 32)) - 1));
}


__device__ float distance( float *sample1, float * sample2, uint32_t
        incr1, uint32_t incr2);


// Currently handled as an array, contingent upon caller to coalesce access.
// sample1 - memory location from which to read the data for point 1. Typically
// in our case going to be the dataset we are looking at.
// sample2 - memory location from which to read the data for point 2. Typicaly
// in our case going to be the centroids we are looking at.
// incr1 - memory jumps to access next feature of point 1. Typically in our case
// is going to be the dataset size.
// incr2 - memory jumps to access next feature of point 2. Typically in our case
// is going to be the number of centroids K.
// num_features - assumed to be in constant memory indicates feature dimension
// space
__device__ float distance( float *sample1, float *sample2, uint32_t incr1,
        uint32_t incr2)
{
    float ret_distance = 0;
    for(int i=0;i<num_features;i++)
        ret_distance +=
            (sample1[i*incr1]-sample2[i*incr2])*(sample1[i*incr1]-sample2[i*incr2]);
    return ret_distance;
}

__global__ void nearest_cluster_assign( float *samples, float *centroids,
        uint32_t *membership, uint32_t *membership_old)
{
    uint32_t sample_idx = blockIdx.x * blockDim.x + threadIdx.x;
    float min_dist = FLT_MAX;
    uint32_t nearest_cluster = num_clusters-1;
    if(sample_idx >= num_samples)
        return;
    // Goto concerned start pointer in samples array
    samples += sample_idx;
    //Create a shared mem buffer for centroid. If all centroids fit into the
    //region, well and good. If not we load it on and off in batches and take
    //distances. Theoretically should benefit from it but yet to see if it
    //actually deteriorates emperically.
    extern __shared__ float shared_centroids[];
    //TODO: Check back to see if this calculation is right or sizeof(float)
    //incorporated elsewhere
    uint32_t max_shared_centroids =
        shmem_size/(num_features*sizeof(float));
    //TODO: define min if needed
    uint32_t nactive_threads = min(blockDim.x, num_samples - blockIdx.x *
            blockDim.x);
    uint32_t thread_num_shared_process = ceilf(max_shared_centroids /
            nactive_threads);
    //Load a batch of centroids to shared and compute pairwise distance between
    //the current point and all centroids
    for(uint32_t centroids_batch=0; centroids_batch<num_clusters;
            centroids_batch += max_shared_centroids)
    {
        for(uint32_t i=0; i<thread_num_shared_process; i++)
        {
            uint32_t local_offset = i * nactive_threads + threadIdx.x;
            uint32_t global_offset = local_offset + centroids_batch;
            //Confused in offsets, put a conditional here to be safe
            if(global_offset<num_clusters && local_offset<max_shared_centroids)
            {
                for(uint32_t feature_idx=0; feature_idx<num_features;
                        feature_idx++)
                {
                    shared_centroids[feature_idx*max_shared_centroids + local_offset] =
                        centroids[feature_idx*num_clusters + global_offset];
                }
            }
        }
        __syncthreads();
        for(uint32_t cluster = 0; cluster < max_shared_centroids && cluster <
                num_clusters - centroids_batch; cluster++)
        {
            float dist = distance(samples, shared_centroids+cluster,
                    num_samples, max_shared_centroids);
            if(dist<min_dist)
            {
                min_dist = dist;
                nearest_cluster = cluster + centroids_batch;
            }
        }
    }
    uint32_t mem_old = membership[sample_idx];
    membership_old[sample_idx] = mem_old;
    if(mem_old != nearest_cluster)
    {
        membership[sample_idx] = nearest_cluster;
        atomicAggInc(&mem_change_ctr);
    }
}

__global__ void adjust_centroids( float *samples, float *centroids, uint32_t
        *membership, uint32_t *membership_old, uint32_t *cluster_counts)
{
    uint32_t centroid_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(centroid_idx >= num_clusters)
        return;
    centroids += centroid_idx;
    uint32_t cluster_count = cluster_counts[centroid_idx];
    //if(d_debug)
    //{
    //    for(uint32_t tmp = 0; tmp < num_features; tmp++)
    //        printf("printing pre-centroid id=%u feature %u: %f with count %u\n", centroid_idx,
    //                tmp, centroids[index(tmp,0,num_clusters)], cluster_count);
    //}

    //multiply each centroid by it's count to make it ready for adjustments -
    //neccessary evil of globalmemory writes
    for(uint32_t i = 0; i < num_features; i++)
    {
        centroids[i*num_clusters] *= cluster_count;
    }
    extern __shared__ uint32_t shared_memberships[];
    //Since each membership is of type uint32_t and we have old and new
    //memberships, we can load the memberships to fill half the shared memory
    uint32_t sample_step = shmem_size/(2*sizeof(uint32_t));
    uint32_t nactive_threads = min(blockDim.x, num_clusters - blockIdx.x *
            blockDim.x);
    uint32_t samples_per_thread = ceilf(sample_step/nactive_threads);
    //Mask off shared mem stuff for now.
    //for(uint32_t sample_start = 0; sample_start < num_samples; sample_start +=
    //        sample_step)
    //{
    //    for(uint32_t i=0; i<samples_per_thread; i++)
    //    {
    //        uint32_t local_offset = i * nactive_threads + threadIdx.x;
    //        uint32_t global_offset = local_offset + sample_start;
    //        if(global_offset < num_samples && local_offset < sample_step)
    //        {
    //            shared_memberships[2*local_offset] = membership[global_offset];
    //            shared_memberships[2*local_offset+1] = membership_old[global_offset];
    //        }
    //    }
    //    __syncthreads();
    //    //Now each thread is going to scan all the shared samples
    //    for(uint32_t i=0; i < sample_step && sample_start + i < num_samples;
    //            i++)
    //    {
    //        uint32_t local_membership = shared_memberships[2*i];
    //        uint32_t local_membership_prev = shared_memberships[2*i+1];
    //        int sign = 0;
    //        if(local_membership_prev == centroid_idx && local_membership !=
    //                centroid_idx)
    //        {
    //            //if(d_debug && cluster_count == 0)
    //            //{
    //            //    printf("Cluster count 0 decrement triggered for cluster=%u,"
    //            //            " on sample %u - membersip changed from %u to %u\n",
    //            //            centroid_idx, sample_start + i,
    //            //            local_membership_prev, local_membership);
    //            //    for(uint32_t tmp = 0; tmp < num_samples; tmp++)
    //            //    {
    //            //        if(membership_old[tmp] == centroid_idx)
    //            //            printf("Cluster %u found for sample %u with chenge"
    //            //                    "to %u\n", centroid_idx, tmp,
    //            //                    membership[tmp]);
    //            //    }
    //            //}
    //            sign = -1;
    //            cluster_count--;
    //        }
    //        else if(local_membership_prev != centroid_idx && local_membership ==
    //                centroid_idx)
    //        {
    //            sign = 1;
    //            cluster_count++;
    //        }
    //        if(sign)
    //        {
    //            uint32_t sample_offset = sample_start + i;
    //            for(uint32_t feature = 0; feature < num_features; feature++)
    //            {
    //                centroids[feature * num_clusters] += sign *
    //                    samples[sample_offset + feature * num_samples];
    //            }
    //        }
    //    }
    //}
    for(uint32_t i = 0; i < num_samples; i++)
    {
        uint32_t local_membership = membership[i];
        uint32_t local_membership_prev = membership_old[i];
        int sign = 0;
        if(local_membership_prev == centroid_idx && local_membership !=
            centroid_idx)
        {
            //if(d_debug && cluster_count == 0)
            //{
            //    printf("Cluster count 0 decrement triggered for cluster=%u,"
            //            " on sample %u - membersip changed from %u to %u\n",
            //            centroid_idx, i,
            //            local_membership_prev, local_membership);
            //    for(uint32_t tmp = 0; tmp < num_samples; tmp++)
            //    {
            //        if(membership_old[tmp] == centroid_idx)
            //            printf("Cluster %u found for sample %u with chenge"
            //                    "to %u\n", centroid_idx, tmp,
            //                    membership[tmp]);
            //    }
            //}
            sign = -1;
            cluster_count--;
        }
        else if(local_membership_prev != centroid_idx && local_membership ==
                centroid_idx)
        {
            sign = 1;
            cluster_count++;
        }
        if(sign)
        {
            for(uint32_t feature = 0; feature < num_features; feature++)
            {
                centroids[feature * num_clusters] += sign *
                    samples[i + feature * num_samples];
            }
        }
    }
    // Average the centroid
    for(uint32_t i = 0; i < num_features; i++)
    {
        //if(d_debug)
        //    printf("printing post-centroid unnormalized id=%u feature %u: %f with count %u\n", 
        //            centroid_idx, i, centroids[index(i,0,num_clusters)], cluster_count);
        centroids[i*num_clusters] /= cluster_count;
    }
    //Write back local count to memory
    cluster_counts[centroid_idx] = cluster_count;
    //if(d_debug)
    //{
    //    for(uint32_t tmp = 0; tmp < num_features; tmp++)
    //        printf("printing post-centroid id=%u feature %u: %f with count %u\n", 
    //                centroid_idx, tmp, centroids[index(tmp,0,num_clusters)],
    //                cluster_count);
    //}
}

//Debugging functions
__global__ void verify_counts(uint32_t *cluster_counts)
{
    uint32_t sum = 0;
    for(uint32_t i = 0; i < num_clusters; i++)
    {
        sum += cluster_counts[i];
    }
    if(sum != num_samples)
        printf("sum of counts (%u) doesn't add up to required (%u)\n", sum,
                num_samples);
}

__global__ void verify_memberships(uint32_t *memberships, uint32_t *cc)
{
    for(uint32_t i = 0; i < num_samples; i++)
    {
        if(memberships[i]<0 || memberships[i]>=num_clusters)
            printf("membership for %u wrongly assigned to %u", i,
                    memberships[i]);
        cc[memberships[i]]++;
    }
}

//------------------------Host Functions--------------------------------

uint32_t initTasks(uint32_t n_samples, uint32_t n_clusters, uint32_t
        n_features, int dev_num=0)
{
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(num_samples), &n_samples, sizeof(n_samples)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(num_clusters), &n_clusters, sizeof(n_clusters)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(num_features), &n_features, sizeof(n_features)));
    hipDeviceProp_t props;
    gpuErrchk(hipSetDevice(dev_num));
    gpuErrchk(hipGetDeviceProperties(&props, dev_num));
    uint32_t smem_size = props.sharedMemPerBlock;
    if(_debug)
        printf("gpu %d has %u bytes of shared memory\n", dev_num, smem_size); 
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(shmem_size), &smem_size, sizeof(smem_size)));
    uint32_t zero = 0;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(mem_change_ctr), &zero, sizeof(zero)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_debug), &_debug, sizeof(_debug)));
    return smem_size;
}

int check_change_ratio(float tolerance, uint32_t n_samples)
{
    uint32_t num_changes = 0;
    gpuErrchk(hipMemcpyFromSymbol(&num_changes, HIP_SYMBOL(mem_change_ctr),
                sizeof(num_changes)));
    if(_debug)
        printf("num changes = %u\n",num_changes);
    if(num_changes <= tolerance * n_samples)
        return -1;
    uint32_t zero = 0;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(mem_change_ctr), &zero, sizeof(zero)));
    return 0;
}

hipError_t kmeans_cuda( InitMethod init, float tolerance, uint32_t n_samples,
        uint32_t n_features, uint32_t n_clusters, uint32_t seed, float
        *samples, float *centroids, uint32_t *memberships, int *iterations =
        NULL)
{
    uint32_t smem_size = initTasks(n_samples, n_clusters, n_features);
    dim3 sample_block(BLOCK_SZ_CNT_ASS);
    dim3 centroid_block(BLOCK_SZ_CNT_ADJ);
    dim3 sample_grid(ceil(1.0 * n_samples/sample_block.x));
    dim3 centroid_grid(ceil(1.0 * n_clusters/centroid_block.x));
    uint32_t *memberships_old, *cluster_counts;
    gpuErrchk(hipMalloc((void **) &memberships_old,
                n_samples*sizeof(uint32_t)));
    gpuErrchk(hipMalloc((void **) &cluster_counts,
                n_clusters*sizeof(uint32_t)));
    uint32_t *cc_verification;
    if(_debug > 1)
    {
        gpuErrchk(hipMalloc((void **) &cc_verification,
                    n_clusters*sizeof(uint32_t)));
        gpuErrchk(hipMemcpy( memberships_old, memberships, n_samples *
                    sizeof(uint32_t), hipMemcpyDeviceToDevice));
    }
    gpuErrchk(hipMemset(cluster_counts, 0, n_clusters*sizeof(uint32_t)));
    //arbitrary - set maxiter to 500
    for(int i = 0; i < 500; i++)
    {
        if(_debug)
        {
            printf("In iteration %d\n",i);
            printf("grid size is %dx%d, block size is %dx%d and shared mem needed is %u\n"
                    , sample_grid.x, sample_grid.y, sample_block.x,
                    sample_block.y, smem_size);
        }
        nearest_cluster_assign<<<sample_grid,sample_block,smem_size>>>( samples,
                centroids, memberships, memberships_old);
        gpuErrchk( hipPeekAtLastError() );
        int change_ratio_good = check_change_ratio(tolerance, n_samples);
        if(_debug)
        {
            printf("change ratio is %d\n",change_ratio_good);
            if(_debug > 1)
            {
                gpuErrchk(hipMemset(cc_verification, 0, n_clusters*sizeof(uint32_t)));
                verify_memberships<<<1,1>>>(memberships, cc_verification);
                verify_counts<<<1,1>>>(cc_verification);
            }
        }
        if(change_ratio_good<0)
        {
            if(iterations)
                *iterations = i;
            return hipSuccess;
        }
        adjust_centroids<<<centroid_grid,centroid_block,smem_size>>>( samples,
                centroids, memberships, memberships_old, cluster_counts);
        gpuErrchk( hipPeekAtLastError() );

        if(_debug > 1)
        {
            verify_counts<<<1,1>>>(cluster_counts);
        }
    }
    return hipSuccess;
}
