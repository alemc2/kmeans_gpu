#include <stdlib.h>
#include <stdio.h>
#include <cmath>

#include "kmeans.h"

hipError_t kmeans_cuda( InitMethod init, float tolerance, uint32_t num_samples,
        uint32_t num_features, uint32_t num_clusters_size, uint32_t seed, const
        float *samples, float *centroids, uint32_t *memberships)
{
    return hipSuccess;
}
