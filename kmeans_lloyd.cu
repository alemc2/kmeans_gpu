#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <float.h>

#include "kmeans.h"

#define BLOCK_SZ_CNT_ASS 256
#define BLOCK_SZ_CNT_ADJ 256

//Define some counters on device for access to changes
__device__ uint32_t mem_change_ctr;
//Define these constant variables that are going to be that way for the entire
//experiment
__constant__ uint32_t num_features;
__constant__ uint32_t num_samples;
__constant__ uint32_t num_clusters;
__constant__ uint32_t shmem_size;

// https://devblogs.nvidia.com/parallelforall/cuda-pro-tip-optimized-filtering-warp-aggregated-atomics/
__device__ __forceinline__ uint32_t atomicAggInc(uint32_t *ctr) {
  int mask = __ballot(1);
  int leader = __ffs(mask) - 1;
  uint32_t res;
  if ((threadIdx.x % 32) == leader) {
    res = atomicAdd(ctr, __popc(mask));
  }
  res = __shfl(res, leader);
  return res + __popc(mask & ((1 << (threadIdx.x % 32)) - 1));
}


__device__ float distance( float *sample1, float * sample2, uint32_t
        incr1, uint32_t incr2);


// Currently handled as an array, contingent upon caller to coalesce access.
// sample1 - memory location from which to read the data for point 1. Typically
// in our case going to be the dataset we are looking at.
// sample2 - memory location from which to read the data for point 2. Typicaly
// in our case going to be the centroids we are looking at.
// incr1 - memory jumps to access next feature of point 1. Typically in our case
// is going to be the dataset size.
// incr2 - memory jumps to access next feature of point 2. Typically in our case
// is going to be the number of centroids K.
// num_features - assumed to be in constant memory indicates feature dimension
// space
__device__ float distance( float *sample1, float *sample2, uint32_t incr1,
        uint32_t incr2)
{
    int ret_distance = 0;
    for(int i=0;i<num_features;i++)
        ret_distance +=
            (sample1[i*incr1]-sample2[i*incr2])*(sample1[i*incr1]-sample2[i*incr2]);
    return ret_distance;
}

__global__ void nearest_cluster_assign( float *samples, float *centroids,
        uint32_t *membership, uint32_t *membership_old)
{
    uint32_t sample_idx = blockIdx.x * blockDim.x + threadIdx.x;
    float min_dist = FLT_MAX;
    uint32_t nearest_cluster = num_clusters-1;
    if(sample_idx >= num_samples)
        return;
    // Goto concerned start pointer in samples array
    samples += sample_idx;
    //Create a shared mem buffer for centroid. If all centroids fit into the
    //region, well and good. If not we load it on and off in batches and take
    //distances. Theoretically should benefit from it but yet to see if it
    //actually deteriorates emperically.
    extern __shared__ float shared_centroids[];
    //TODO: Check back to see if this calculation is right or sizeof(float)
    //incorporated elsewhere
    const uint32_t max_shared_centroids =
        shmem_size/(num_features*sizeof(float));
    //TODO: define min if needed
    const uint32_t thread_num_shared_process =
        ceilf(max_shared_centroids/min(blockDim.x, num_samples - blockIdx.x *
                blockDim.x));
    //Load a batch of centroids to shared and compute pairwise distance between
    //the current point and all centroids
    for(uint32_t centroids_batch=0; centroids_batch<num_clusters;
            centroids_batch += max_shared_centroids)
    {
        for(uint32_t i=0; i<thread_num_shared_process; i++)
        {
            uint32_t local_offset = i * thread_num_shared_process + threadIdx.x;
            uint32_t global_offset = local_offset + centroids_batch;
            //Confused in offsets, put a conditional here to be safe
            if(global_offset<num_clusters && local_offset<max_shared_centroids)
            {
                for(uint32_t feature_idx=0; feature_idx<num_features;
                        feature_idx++)
                {
                    shared_centroids[feature_idx*num_clusters + local_offset] =
                        centroids[feature_idx*num_clusters + global_offset];
                }
            }
        }
        __syncthreads();
        for(uint32_t cluster = centroids_batch; cluster < centroids_batch +
                max_shared_centroids && cluster < num_clusters; cluster++)
        {
            float dist = distance(samples, shared_centroids+cluster,
                    num_samples, max_shared_centroids);
            if(dist<min_dist)
            {
                min_dist = dist;
                nearest_cluster = cluster;
            }
        }
    }
    uint32_t mem_old = membership[sample_idx];
    membership_old[sample_idx] = mem_old;
    if(mem_old != nearest_cluster)
    {
        membership[sample_idx] = nearest_cluster;
        atomicAggInc(&mem_change_ctr);
    }
}

__global__ void adjust_centroids( float *samples, float *centroids, uint32_t
        *membership, uint32_t *membership_old, uint32_t *cluster_counts)
{
    uint32_t centroid_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(centroid_idx >= num_clusters)
        return;
    centroids += centroid_idx;
    uint32_t cluster_count = cluster_counts[centroid_idx];
    //multiply each centroid by it's count to make it ready for adjustments -
    //neccessary evil of globalmemory writes
    for(uint32_t i = 0; i < num_features; i++)
    {
        centroids[i*num_clusters] *= cluster_count;
    }
    extern __shared__ uint32_t shared_memberships[];
    //Since each membership is of type uint32_t and we have old and new
    //memberships, we can load the memberships to fill half the shared memory
    uint32_t sample_step = shmem_size/(2*sizeof(uint32_t));
    uint32_t samples_per_thread = ceilf(sample_step/min(blockDim.x, num_clusters
                - blockDim.x * blockDim.x));
    for(uint32_t sample_start = 0; sample_start < num_samples; sample_start +=
            sample_step)
    {
        for(uint32_t i=0; i<samples_per_thread; i++)
        {
            uint32_t local_offset = i * samples_per_thread + threadIdx.x;
            uint32_t global_offset = local_offset + sample_start;
            if(global_offset < num_samples && local_offset < sample_step)
            {
                shared_memberships[2*local_offset] = membership[global_offset];
                shared_memberships[2*local_offset+1] = membership_old[global_offset];
            }
        }
        __syncthreads();
        //Now each thread is going to scan all the shared samples
        for(uint32_t i=0; i < sample_step && sample_start + i < num_samples;
                i++)
        {
            uint32_t local_membership = shared_memberships[2*i];
            uint32_t local_membership_prev = shared_memberships[2*i+1];
            int sign = 0;
            if(local_membership_prev == centroid_idx && local_membership !=
                    centroid_idx)
            {
                sign = -1;
                cluster_count--;
            }
            else if(local_membership_prev != centroid_idx && local_membership ==
                    centroid_idx)
            {
                sign = 1;
                cluster_count++;
            }
            if(sign)
            {
                uint32_t sample_offset = sample_start + i;
                for(uint32_t feature = 0; feature < num_features; feature++)
                {
                    centroids[feature * num_clusters] += sign *
                        samples[sample_offset + feature * num_samples];
                }
            }
        }
    }
    // Average the centroid
    for(uint32_t i = 0; i < num_features; i++)
        centroids[i*num_clusters] /= cluster_count;
    //Write back local count to memory
    cluster_counts[centroid_idx] = cluster_count;
}

//------------------------Host Functions--------------------------------

uint32_t initTasks(uint32_t n_samples, uint32_t n_clusters, uint32_t
        n_features, int dev_num=0)
{
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(num_samples), &n_samples, sizeof(n_samples)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(num_clusters), &n_clusters, sizeof(n_clusters)));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(num_features), &n_features, sizeof(n_features)));
    hipDeviceProp_t props;
    gpuErrchk(hipSetDevice(dev_num));
    gpuErrchk(hipGetDeviceProperties(&props, dev_num));
    uint32_t smem_size = props.sharedMemPerBlock;
    printf("gpu %d has %d bytes of shared memory\n", dev_num, smem_size); 
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(shmem_size), &smem_size, sizeof(smem_size)));
    uint32_t zero = 0;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(mem_change_ctr), &zero, sizeof(zero)));
    return smem_size;
}

int check_change_ratio(float tolerance, uint32_t n_samples)
{
    uint32_t num_changes = 0;
    gpuErrchk(hipMemcpyFromSymbol(&num_changes, HIP_SYMBOL(mem_change_ctr),
                sizeof(num_changes)));
    printf("numchanges = %d\n",num_changes);
    if(num_changes <= tolerance * n_samples)
        return -1;
    uint32_t zero = 0;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(mem_change_ctr), &zero, sizeof(zero)));
    return 0;
}

hipError_t kmeans_cuda( InitMethod init, float tolerance, uint32_t n_samples,
        uint32_t n_features, uint32_t n_clusters, uint32_t seed, float
        *samples, float *centroids, uint32_t *memberships, int *iterations =
        NULL)
{
    uint32_t smem_size = initTasks(n_samples, n_clusters, n_features);
    dim3 sample_block(BLOCK_SZ_CNT_ASS);
    dim3 centroid_block(BLOCK_SZ_CNT_ADJ);
    dim3 sample_grid(ceil(1.0 * n_samples/sample_block.x));
    dim3 centroid_grid(ceil(1.0 * n_clusters/centroid_block.x));
    uint32_t *memberships_old, *cluster_counts;
    gpuErrchk(hipMalloc((void **) &memberships_old,
                n_samples*sizeof(uint32_t)));
    gpuErrchk(hipMalloc((void **) &cluster_counts,
                n_clusters*sizeof(uint32_t)));
    gpuErrchk(hipMemset(cluster_counts, 0, n_clusters*sizeof(uint32_t)));
    //arbitrary - set maxiter to 500
    for(int i = 0; i < 500; i++)
    {
        printf("grid size is %dx%d, block size is %dx%d and shared mem needed is %d\n"
                , sample_grid.x, sample_grid.y, sample_block.x,
                sample_block.y, smem_size);
        nearest_cluster_assign<<<sample_grid,sample_block,smem_size>>>( samples,
                centroids, memberships, memberships_old);
        gpuErrchk( hipPeekAtLastError() );
        int change_ratio_good = check_change_ratio(tolerance, n_samples);
        printf("change ratio is %d\n",change_ratio_good);
        if(change_ratio_good<0)
        {
            if(iterations)
                *iterations = i;
            return hipSuccess;
        }
        adjust_centroids<<<centroid_grid,centroid_block,smem_size>>>( samples,
                centroids, memberships, memberships_old, cluster_counts);
        gpuErrchk( hipPeekAtLastError() );
    }
    return hipSuccess;
}
