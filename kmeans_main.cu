#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <time.h>

#include <fstream>
#include <iostream>
#include <cmath>

#include "kmeans.h"

using namespace std;

int _debug;

void usage(char *argv0, float threshold) {
    char *help =
        "Usage: %s [switches] -i filename -n num_clusters\n"
        "       -i filename    : file containing data to be clustered\n"
        "       -b             : input file is in binary format (default no)- if \
        cluster file provided it should be in same format\n"
        "       -n num_clusters: number of clusters (K must > 1)\n"
        "       -t threshold   : threshold value (default %.4f)\n"
        "       -c clusters    : file containg clusters to initialize to\n"
        "       -d             : enable debug mode\n";
    fprintf(stderr, help, argv0, threshold);
    exit(-1);
}

int main(int argc, char * argv[])
{
    int opt;
    extern char *optarg;
    extern int optind;
    int isBinaryFile;
    Init_Method cluster_method;
    char *cluster_file;
    uint32_t numClusters, numFeatures, numSamples;
    uint32_t numClusters_read,numCluster_Features_read;
    uint32_t *membership;
    char *filename;
    float **samples;
    float *samples_T;
    float *clusters;
    float **clusters_2d;
    float *clusters_T;
    float *d_samples, *d_clusters;
    uint32_t *d_memberships;
    int numIterations;
    float threshold;
    uint32_t seed;
    
    /* some default values */
    _debug           = 0;
    threshold        = 0.001;
    numClusters      = 0;
    isBinaryFile     = 0;
    cluster_method   = InitMethodRandom;
    cluster_file     = NULL;
    filename         = NULL;

    while ( (opt=getopt(argc,argv,"i:n:t:c:d:abo"))!= EOF) {
        switch (opt) {
            case 'i': filename=optarg;
                      break;
            case 'c': cluster_method = InitMethodImport;
                      cluster_file = optarg;
                      break;
            case 'b': isBinaryFile = 1;
                      break;
            case 't': threshold=atof(optarg);
                      break;
            case 'n': numClusters = atoi(optarg);
                      break;
            case 'd': _debug = atoi(optarg);
                      break;
            case '?': usage(argv[0], threshold);
                      break;
            default: usage(argv[0], threshold);
                      break;
        }
    }

    if (filename == 0 || numClusters <= 1) usage(argv[0], threshold);
    if (cluster_method == InitMethodImport && cluster_file == 0) usage(argv[0], threshold);

    samples = file_read(isBinaryFile, filename, &numSamples, &numFeatures);
    if(samples == NULL) exit(1);

    membership = (uint32_t *) malloc(numSamples * sizeof(uint32_t));
    assert(membership != NULL);

    memset(membership, 255, numSamples
             * sizeof(uint32_t));

    samples_T = transpose(samples[0], numSamples, numFeatures);
    seed = time(NULL);
    if (cluster_method == InitMethodImport)
    {
        clusters_2d = file_read(isBinaryFile, cluster_file, &numClusters_read,
                &numCluster_Features_read);
        if(clusters_2d == NULL)
        {
            fprintf(stderr, "Invalid cluster file %s\n", cluster_file);
            exit(1);
        }
        if(numClusters_read != numClusters || numCluster_Features_read !=
                numFeatures)
        {
            fprintf(stderr, "Cluster sizes don't match provided inputs\n");
            exit(1);
        }
        clusters = transpose(clusters_2d[0], numClusters, numFeatures);
        free(clusters_2d[0]);
        free(clusters_2d);
    }
    else
    {
        clusters = (float *) malloc(numClusters * numFeatures * sizeof(float));
    }
    init_centroids( cluster_method, numSamples, numFeatures, numClusters,
            seed, samples_T, clusters);
    if(_debug)
    {
        printf("init clusters are as follows:\n");
        print2d(clusters,numFeatures,numClusters);
    }

    //GPU part
    gpuErrchk(hipMalloc((void **) &d_samples, numSamples * numFeatures *
                sizeof(float)));
    gpuErrchk(hipMalloc((void **) &d_clusters, numClusters * numFeatures *
                sizeof(float)));
    gpuErrchk(hipMalloc((void **) &d_memberships, numSamples *
                sizeof(uint32_t)));
    gpuErrchk(hipMemcpy(d_samples, samples_T, numSamples * numFeatures *
                sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_clusters, clusters, numClusters * numFeatures *
                sizeof(float), hipMemcpyHostToDevice));
    if(_debug > 1)
    {
        printf("printing cpu mem\n");
        print1d(membership,numSamples);
    }
    gpuErrchk(hipMemcpy(d_memberships, membership, numSamples *
                sizeof(uint32_t), hipMemcpyHostToDevice));
    
    kmeans_cuda( InitMethodRandom, threshold, numSamples, numFeatures,
            numClusters, seed, d_samples, d_clusters, d_memberships,
            &numIterations);

    gpuErrchk(hipMemcpy(clusters, d_clusters, numClusters * numFeatures *
                sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(membership, d_memberships, numSamples *
                sizeof(uint32_t), hipMemcpyDeviceToHost));

    clusters_T = transpose(clusters, numFeatures, numClusters);
    clusters_2d = (float**)malloc(numClusters * sizeof(float*));
    for(uint32_t i = 0; i < numClusters; i++)
        clusters_2d[i] = clusters_T + i * numFeatures;
    if(_debug)
    {
        printf("post cluster centroids are:\n");
        print2d(clusters,numFeatures,numClusters);
    }
    file_write(filename, numClusters, numSamples, numFeatures, clusters_2d,
            membership);
    printf("It ran %d number of iterations\n", numIterations);
    return 0;
}
